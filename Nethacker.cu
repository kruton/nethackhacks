#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cutil.h"

#define VECSIZE 31
#define FIRST_USABLE 344
#define USABLE_TOTAL 16
#define POOL_SIZE (FIRST_USABLE + USABLE_TOTAL)
#define NUM_THREADS 256
#define NUM_BLOCKS 128

#define MAX_HITS 32

#define MAX_MATCH_ATTEMPTS 3

#define MAX_ITER 0xFFFFFFFF
#define NUM_PER_ITER (NUM_THREADS * NUM_BLOCKS)
#define TOTAL_ITER (MAX_ITER / NUM_PER_ITER)

#define MUL(a, b) __umul24(a, b)

#define RND(a) (((unsigned int)buffer[a]) >> 1)

typedef struct {
	unsigned int fact[VECSIZE];
} intvec;

__global__ void attack(const unsigned int seed, const intvec *pool, const int *engraving, const int engraving_length, const int *offsets, const int *changes, const int num_changes, unsigned int *hits, unsigned int *hit_number) {
	unsigned int idx = MUL(blockIdx.x, blockDim.x) + threadIdx.x;
	int buffer[POOL_SIZE];

	long int word = seed + idx;
	buffer[0] = word;

	/* Initial seed of the RNG for period 31 */
	#pragma unroll
	for (int i = 1; i < 31; ++i) {
		long int hi = word / 127773;
		long int lo = word % 127773;
		word = MUL(16807, lo) - MUL(2836, hi);
		if (word < 0)
			word += 2147483647;
		buffer[i] = word;
	}

	/* Finish up RNG state */
	#pragma unroll
	for (int i = 31; i < 34; ++i) {
		buffer[i] = buffer[i - 31];
	}

	/* Generate numbers to discard plus USABLE_TOTAL real numbers */
	#pragma unroll
	for (int i = 34; i < POOL_SIZE; ++i) {
		buffer[i] = buffer[i - 31] + buffer[i - 3];
	}

	/* Start with the first usable number */
	int randidx = FIRST_USABLE;

	int letter;
	int offset;
	int attempt = 0;
	int i = 0;
	for (i = 0; i < num_changes && randidx < POOL_SIZE; ++i) {
		do offset = RND(randidx++) % engraving_length;
			while (offset != offsets[i] && ++attempt < MAX_MATCH_ATTEMPTS);
		if (attempt == MAX_MATCH_ATTEMPTS)
			return;
		do letter = RND(randidx++) % 25;
			while (letter == engraving[offsets[i]]);
		if (letter != changes[i])
			return;
	}

	unsigned int hit_location = atomicAdd(hit_number, 1);
	if (hit_location < MAX_HITS)
		hits[hit_location] = buffer[0];
}

void
init(int offset, intvec *pool)
{
    int kc = VECSIZE * 10 + offset;

    for (int i = 0; i < VECSIZE; ++i)
        pool[i].fact[i] = 1;

    int f = 3; int r = 0;
    while (--kc >= 0) {
        for (int i = 0; i < 31; ++i)
            pool[f].fact[i] += pool[r].fact[i];
        ++f;
        if (f >= VECSIZE) {
            f = 0;
            ++r;
        } else {
            ++r;
            if (r >= VECSIZE)
              r = 0;
        }
    }
}

long int find_seed(const char *engraving, int engraving_length, const int *offsets, const int *changes, int num_changes) {
	dim3 threads, blocks;
	float time_kernel;

	int cuda_device = 0;
	int num_devices = 0;
	CUDA_SAFE_CALL( hipGetDeviceCount(&num_devices) );

	if (num_devices == 0) {
		printf("Your system does not have a CUDA-capable device.\n");
		return 1;
	}

	if (cuda_device >= num_devices) {
		printf("Choose device ID between 0 and %d.\n", num_devices - 1);
		return 1;
	}

	hipSetDevice(cuda_device);

	hipEvent_t start_event, stop_event;
	CUDA_SAFE_CALL( hipEventCreate(&start_event) );
	CUDA_SAFE_CALL( hipEventCreate(&stop_event) );

	/* Convert engraving to integers */
	int *eng_ints = (int *)malloc(sizeof(int) * engraving_length);
	for (int i = 0; i < engraving_length; ++i)
		eng_ints[i] = engraving[i] - 'a';
	
	/* Allocate device buffers */
	int *engraving_d, *changes_d;
	int *offsets_d;
	intvec *pool_d;
	unsigned int *hits_d, *hit_number_d;
	CUDA_SAFE_CALL( hipMalloc((void**)&engraving_d, sizeof(int) * engraving_length) );
	CUDA_SAFE_CALL( hipMalloc((void**)&offsets_d, sizeof(int) * num_changes) );
	CUDA_SAFE_CALL( hipMalloc((void**)&changes_d, sizeof(int) * num_changes) );
	CUDA_SAFE_CALL( hipMalloc((void**)&hits_d, sizeof(unsigned int) * MAX_HITS) );
	CUDA_SAFE_CALL( hipMalloc((void**)&hit_number_d, sizeof(unsigned int)) );
	CUDA_SAFE_CALL( hipMalloc((void**)&pool_d, sizeof(intvec) * VECSIZE) );

	/* Set up parameters of kernel run */
	threads = dim3(NUM_THREADS, 1);
	blocks = dim3(NUM_BLOCKS, 1);

	/* seed initial pool data */
	intvec pool[31];
	init(2, pool);

	/* initialize hit data */
	unsigned int hit_number = 0;
	unsigned int hits[MAX_HITS];
	unsigned long seed = 0;

	/* Copy all data from host to device */
	printf("Copying data from host to CUDA device...\n");
	hipMemcpy(engraving_d, eng_ints, sizeof(int) * engraving_length, hipMemcpyHostToDevice);
	hipMemcpy(offsets_d, offsets, sizeof(int) * num_changes, hipMemcpyHostToDevice);
	hipMemcpy(changes_d, changes, sizeof(int) * num_changes, hipMemcpyHostToDevice);
	hipMemcpy(hits_d, &hits, sizeof(unsigned int) * MAX_HITS, hipMemcpyHostToDevice);
	hipMemcpy(hit_number_d, &hit_number, sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(pool_d, &pool, sizeof(intvec) * VECSIZE, hipMemcpyHostToDevice);

	/* Print out initial status bar for user to see. */
	printf("Starting seed search on CUDA device...\n");
	printf("[");
	for (int i = 0; i < 50; ++i)
		printf(".");
	printf("] %3u%%", 0);
	fflush(stdout);

	/* Run kernel in loop to prevent GUI from becoming unresponsive */
	hipEventRecord(start_event, 0);
	const int report_after = TOTAL_ITER / 100;
	int reports = report_after;
	for (int i = 0; i < TOTAL_ITER && !hit_number; ++i) {
		attack<<<threads, blocks>>>(seed, pool_d, engraving_d, engraving_length, offsets_d, changes_d, num_changes, hits_d, hit_number_d);
		CUT_CHECK_ERROR("Kernel execution failed");
		hipMemcpy(&hit_number, hit_number_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
		if (hit_number > 0)
			hipMemcpy(&hits, hits_d, sizeof(unsigned int) * MAX_HITS, hipMemcpyDeviceToHost);
		seed += NUM_PER_ITER;
		if (--reports == 0) {
			unsigned int pct_complete = (seed * 100LL) / MAX_ITER;
			unsigned int blocks_done = pct_complete / 2;
			printf("\r[");
			for (int pct = 0; pct < blocks_done; ++pct)
				printf("#");
			for (int pct = blocks_done; pct < 50; ++pct)
				printf(".");
			printf("] %3u%%", pct_complete);
			fflush(stdout);
			reports = report_after;
		}
	}
	hipEventRecord(stop_event, 0);
	hipEventSynchronize(stop_event);
	CUDA_SAFE_CALL( hipEventElapsedTime(&time_kernel, start_event, stop_event) );
	printf("\n\nFinished scan of space in %.2f ms (%.2f seeds/ms)\n", time_kernel, seed / time_kernel);

	printf("number of hits = %u, last seed = %u\n", hit_number, seed);
	if (hit_number > 0) {
		printf("Seed hits:\n");
		for (int i = 0; i < min(hit_number, MAX_HITS); ++i) {
			printf("\t%u\n", hits[i]);
		}
	}

	/* Tear down for exit */
	hipEventDestroy(start_event);
	hipEventDestroy(stop_event);

	CUDA_SAFE_CALL( hipFree((void**)&engraving_d) );
	CUDA_SAFE_CALL( hipFree((void**)&offsets_d) );
	CUDA_SAFE_CALL( hipFree((void**)&changes_d) );
	CUDA_SAFE_CALL( hipFree((void**)&hits_d) );
	CUDA_SAFE_CALL( hipFree((void**)&hit_number_d) );

	free(eng_ints);

	return hit_number;
}
